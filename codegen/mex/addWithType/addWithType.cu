//
//  addWithType.cu
//
//  Code generation for function 'addWithType'
//


// Include files
#include "addWithType.h"
#include "_coder_addWithType_mex.h"
#include "rt_nonfinite.h"

// Variable Definitions
emlrtCTX emlrtRootTLSGlobal = NULL;
emlrtContext emlrtContextGlobal = { true,// bFirstTime
  false,                               // bInitialized
  131594U,                             // fVersionInfo
  NULL,                                // fErrorFunction
  "addWithType",                       // fFunctionName
  NULL,                                // fRTCallStack
  false,                               // bDebugMode
  { 3194014219U, 53452778U, 2865749887U, 3590888462U },// fSigWrd
  NULL                                 // fSigMem
};

// Function Declarations
static real_T b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId);
static real_T c_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier
  *msgId);
static real_T emlrt_marshallIn(const mxArray *a, const char_T *identifier);
static const mxArray *emlrt_marshallOut(const real_T u);

// Function Definitions
static real_T b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId)
{
  real_T y;
  y = c_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static real_T c_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier
  *msgId)
{
  real_T ret;
  static const int32_T dims = 0;
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "double", false, 0U,
    &dims);
  ret = *(real_T *)emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static real_T emlrt_marshallIn(const mxArray *a, const char_T *identifier)
{
  real_T y;
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char *>(identifier);
  thisId.fParent = NULL;
  thisId.bParentIsCell = false;
  y = b_emlrt_marshallIn(emlrtAlias(a), &thisId);
  emlrtDestroyArray(&a);
  return y;
}

static const mxArray *emlrt_marshallOut(const real_T u)
{
  const mxArray *y;
  const mxArray *m;
  y = NULL;
  m = emlrtCreateDoubleScalar(u);
  emlrtAssign(&y, m);
  return y;
}

real_T addWithType(real_T a, real_T b)
{
  return a + b;
}

void addWithType_api(const mxArray * const prhs[2], int32_T, const mxArray *
                     plhs[1])
{
  real_T a;
  real_T b;

  // Marshall function inputs
  a = emlrt_marshallIn(emlrtAliasP(prhs[0]), "a");
  b = emlrt_marshallIn(emlrtAliasP(prhs[1]), "b");

  // Invoke the target function
  // Marshall function outputs
  plhs[0] = emlrt_marshallOut(addWithType(a, b));
}

void addWithType_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void addWithType_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, 0);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, "Distrib_Computing_Toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  hipGetLastError();
}

void addWithType_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(false, emlrtRootTLSGlobal);
  }

  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (addWithType.cu)
