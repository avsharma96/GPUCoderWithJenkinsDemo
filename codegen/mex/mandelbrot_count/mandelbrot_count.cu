#include "hip/hip_runtime.h"
//
//  mandelbrot_count.cu
//
//  Code generation for function 'mandelbrot_count'
//


// Include files
#include "mandelbrot_count.h"
#include "MWCudaDimUtility.hpp"
#include "rt_nonfinite.h"

// Function Declarations
static __global__ void mandelbrot_count_kernel1(const real_T yGrid[1000000],
  const real_T xGrid[1000000], creal_T z[1000000], real_T count[1000000],
  creal_T z0[1000000]);
static __global__ void mandelbrot_count_kernel2(const creal_T z0[1000000],
  real_T count[1000000], creal_T z[1000000]);
static __global__ void mandelbrot_count_kernel3(real_T count[1000000]);
static __device__ real_T rt_hypotd_snf(real_T u0, real_T u1);

// Function Definitions
static __global__ __launch_bounds__(512, 1) void mandelbrot_count_kernel1(const
  real_T yGrid[1000000], const real_T xGrid[1000000], creal_T z[1000000], real_T
  count[1000000], creal_T z0[1000000])
{
  uint32_T threadId;
  int32_T k;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  k = static_cast<int32_T>(threadId);
  if (k < 1000000) {
    real_T re;

    //  Copyright 2016-2019 The MathWorks, Inc.
    //  Map computation to GPU.
    re = xGrid[k] + 0.0 * yGrid[k];
    z0[k].re = re;
    z0[k].im = yGrid[k];
    count[k] = 1.0;
    z[k].re = re;
    z[k].im = yGrid[k];
  }
}

static __global__ __launch_bounds__(512, 1) void mandelbrot_count_kernel2(const
  creal_T z0[1000000], real_T count[1000000], creal_T z[1000000])
{
  uint32_T threadId;
  int32_T k;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  k = static_cast<int32_T>(threadId);
  if (k < 1000000) {
    real_T re;
    real_T im;
    re = (z[k].re * z[k].re - z[k].im * z[k].im) + z0[k].re;
    im = (z[k].re * z[k].im + z[k].im * z[k].re) + z0[k].im;
    z[k].re = re;
    z[k].im = im;
    count[k] += static_cast<real_T>(rt_hypotd_snf(re, im) <= 2.0);
  }
}

static __global__ __launch_bounds__(512, 1) void mandelbrot_count_kernel3(real_T
  count[1000000])
{
  uint32_T threadId;
  int32_T k;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  k = static_cast<int32_T>(threadId);
  if (k < 1000000) {
    count[k] = log(count[k]);
  }
}

static __device__ real_T rt_hypotd_snf(real_T u0, real_T u1)
{
  real_T y;
  real_T a;
  a = fabs(u0);
  y = fabs(u1);
  if (a < y) {
    a /= y;
    y *= sqrt(a * a + 1.0);
  } else if (a > y) {
    y /= a;
    y = a * sqrt(y * y + 1.0);
  } else {
    if (!static_cast<int32_T>(isnan(y))) {
      y = a * 1.4142135623730951;
    }
  }

  return y;
}

void mandelbrot_count(real_T maxIterations, const real_T xGrid[1000000], const
                      real_T yGrid[1000000], real_T count[1000000])
{
  int32_T i;
  real_T (*gpu_yGrid)[1000000];
  real_T (*gpu_xGrid)[1000000];
  creal_T (*gpu_z)[1000000];
  real_T (*gpu_count)[1000000];
  creal_T (*gpu_z0)[1000000];
  hipMalloc(&gpu_z0, 16000000ULL);
  hipMalloc(&gpu_count, 8000000ULL);
  hipMalloc(&gpu_z, 16000000ULL);
  hipMalloc(&gpu_xGrid, 8000000ULL);
  hipMalloc(&gpu_yGrid, 8000000ULL);

  //  Copyright 2016-2019 The MathWorks, Inc.
  //  Map computation to GPU.
  hipMemcpy(gpu_yGrid, (void *)&yGrid[0], 8000000ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_xGrid, (void *)&xGrid[0], 8000000ULL, hipMemcpyHostToDevice);
  mandelbrot_count_kernel1<<<dim3(1954U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_yGrid, *gpu_xGrid, *gpu_z, *gpu_count, *gpu_z0);
  i = static_cast<int32_T>(maxIterations + 1.0);
  for (int32_T n = 0; n < i; n++) {
    mandelbrot_count_kernel2<<<dim3(1954U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_z0, *gpu_count, *gpu_z);
  }

  mandelbrot_count_kernel3<<<dim3(1954U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_count);
  hipMemcpy(&count[0], gpu_count, 8000000ULL, hipMemcpyDeviceToHost);
  hipFree(*gpu_yGrid);
  hipFree(*gpu_xGrid);
  hipFree(*gpu_z);
  hipFree(*gpu_count);
  hipFree(*gpu_z0);
}

// End of code generation (mandelbrot_count.cu)
