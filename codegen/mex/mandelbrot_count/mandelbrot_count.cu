#include "hip/hip_runtime.h"
//
//  mandelbrot_count.cu
//
//  Code generation for function 'mandelbrot_count'
//


// Include files
#include "mandelbrot_count.h"
#include "MWCudaDimUtility.hpp"
#include "_coder_mandelbrot_count_mex.h"
#include "rt_nonfinite.h"

// Variable Definitions
emlrtCTX emlrtRootTLSGlobal = NULL;
emlrtContext emlrtContextGlobal = { true,// bFirstTime
  false,                               // bInitialized
  131594U,                             // fVersionInfo
  NULL,                                // fErrorFunction
  "mandelbrot_count",                  // fFunctionName
  NULL,                                // fRTCallStack
  false,                               // bDebugMode
  { 3194014219U, 53452778U, 2865749887U, 3590888462U },// fSigWrd
  NULL                                 // fSigMem
};

// Function Declarations
static real_T b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId);
static real_T (*c_emlrt_marshallIn(const mxArray *xGrid, const char_T
  *identifier))[1000000];
static real_T (*d_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId))[1000000];
static real_T e_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier
  *msgId);
static real_T emlrt_marshallIn(const mxArray *maxIterations, const char_T
  *identifier);
static const mxArray *emlrt_marshallOut(const real_T u[1000000]);
static real_T (*f_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier *
  msgId))[1000000];
static __global__ void mandelbrot_count_kernel1(const real_T yGrid[1000000],
  const real_T xGrid[1000000], creal32_T z[1000000], real_T count[1000000],
  creal32_T z0[1000000]);
static __global__ void mandelbrot_count_kernel2(const creal32_T z0[1000000],
  real_T count[1000000], creal32_T z[1000000]);
static __global__ void mandelbrot_count_kernel3(real_T count[1000000]);
static __device__ real32_T rt_hypotf_snf(real32_T u0, real32_T u1);

// Function Definitions
static real_T b_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier
  *parentId)
{
  real_T y;
  y = e_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static real_T (*c_emlrt_marshallIn(const mxArray *xGrid, const char_T
  *identifier))[1000000]
{
  real_T (*y)[1000000];
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char *>(identifier);
  thisId.fParent = NULL;
  thisId.bParentIsCell = false;
  y = d_emlrt_marshallIn(emlrtAlias(xGrid), &thisId);
  emlrtDestroyArray(&xGrid);
  return y;
}
  static real_T (*d_emlrt_marshallIn(const mxArray *u, const emlrtMsgIdentifier *
  parentId))[1000000]
{
  real_T (*y)[1000000];
  y = f_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static real_T e_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier
  *msgId)
{
  real_T ret;
  static const int32_T dims = 0;
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "double", false, 0U,
    &dims);
  ret = *(real_T *)emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static real_T emlrt_marshallIn(const mxArray *maxIterations, const char_T
  *identifier)
{
  real_T y;
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char *>(identifier);
  thisId.fParent = NULL;
  thisId.bParentIsCell = false;
  y = b_emlrt_marshallIn(emlrtAlias(maxIterations), &thisId);
  emlrtDestroyArray(&maxIterations);
  return y;
}

static const mxArray *emlrt_marshallOut(const real_T u[1000000])
{
  const mxArray *y;
  const mxArray *m;
  static const int32_T iv[2] = { 0, 0 };

  static const int32_T iv1[2] = { 1000, 1000 };

  y = NULL;
  m = emlrtCreateNumericArray(2, &iv[0], mxDOUBLE_CLASS, mxREAL);
  emlrtMxSetData((mxArray *)m, (void *)&u[0]);
  emlrtSetDimensions((mxArray *)m, *(int32_T (*)[2])&iv1[0], 2);
  emlrtAssign(&y, m);
  return y;
}

static real_T (*f_emlrt_marshallIn(const mxArray *src, const emlrtMsgIdentifier *
  msgId))[1000000]
{
  real_T (*ret)[1000000];
  static const int32_T dims[2] = { 1000, 1000 };

  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "double", false, 2U,
    dims);
  ret = (real_T (*)[1000000])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}
  static __global__ __launch_bounds__(512, 1) void mandelbrot_count_kernel1(
  const real_T yGrid[1000000], const real_T xGrid[1000000], creal32_T z[1000000],
  real_T count[1000000], creal32_T z0[1000000])
{
  uint32_T threadId;
  int32_T k;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  k = static_cast<int32_T>(threadId);
  if (k < 1000000) {
    real32_T re;

    //  Copyright 2016-2019 The MathWorks, Inc.
    //  Map computation to GPU.
    re = static_cast<real32_T>(xGrid[k] + 0.0 * yGrid[k]);
    z0[k].re = re;
    z0[k].im = static_cast<real32_T>(yGrid[k]);
    count[k] = 1.0;
    z[k].re = re;
    z[k].im = static_cast<real32_T>(yGrid[k]);
  }
}

static __global__ __launch_bounds__(512, 1) void mandelbrot_count_kernel2(const
  creal32_T z0[1000000], real_T count[1000000], creal32_T z[1000000])
{
  uint32_T threadId;
  int32_T k;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  k = static_cast<int32_T>(threadId);
  if (k < 1000000) {
    real32_T re;
    real32_T im;
    re = (z[k].re * z[k].re - z[k].im * z[k].im) + z0[k].re;
    im = (z[k].re * z[k].im + z[k].im * z[k].re) + z0[k].im;
    z[k].re = re;
    z[k].im = im;
    count[k] += static_cast<real_T>(rt_hypotf_snf(re, im) <= 2.0F);
  }
}

static __global__ __launch_bounds__(512, 1) void mandelbrot_count_kernel3(real_T
  count[1000000])
{
  uint32_T threadId;
  int32_T k;
  threadId = static_cast<uint32_T>(mwGetGlobalThreadIndex());
  k = static_cast<int32_T>(threadId);
  if (k < 1000000) {
    count[k] = log(count[k]);
  }
}

static __device__ real32_T rt_hypotf_snf(real32_T u0, real32_T u1)
{
  real32_T y;
  real32_T a;
  a = fabsf(u0);
  y = fabsf(u1);
  if (a < y) {
    a /= y;
    y *= sqrtf(a * a + 1.0F);
  } else if (a > y) {
    y /= a;
    y = a * sqrtf(y * y + 1.0F);
  } else {
    if (!static_cast<int32_T>(isnan(y))) {
      y = a * 1.41421354F;
    }
  }

  return y;
}

void mandelbrot_count(real_T maxIterations, const real_T xGrid[1000000], const
                      real_T yGrid[1000000], real_T count[1000000])
{
  int32_T i;
  real_T (*gpu_yGrid)[1000000];
  real_T (*gpu_xGrid)[1000000];
  creal32_T (*gpu_z)[1000000];
  real_T (*gpu_count)[1000000];
  creal32_T (*gpu_z0)[1000000];
  hipMalloc(&gpu_z0, 8000000ULL);
  hipMalloc(&gpu_count, 8000000ULL);
  hipMalloc(&gpu_z, 8000000ULL);
  hipMalloc(&gpu_xGrid, 8000000ULL);
  hipMalloc(&gpu_yGrid, 8000000ULL);

  //  Copyright 2016-2019 The MathWorks, Inc.
  //  Map computation to GPU.
  hipMemcpy(gpu_yGrid, (void *)&yGrid[0], 8000000ULL, hipMemcpyHostToDevice);
  hipMemcpy(gpu_xGrid, (void *)&xGrid[0], 8000000ULL, hipMemcpyHostToDevice);
  mandelbrot_count_kernel1<<<dim3(1954U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_yGrid, *gpu_xGrid, *gpu_z, *gpu_count, *gpu_z0);
  i = static_cast<int32_T>(maxIterations + 1.0);
  for (int32_T n = 0; n < i; n++) {
    mandelbrot_count_kernel2<<<dim3(1954U, 1U, 1U), dim3(512U, 1U, 1U)>>>
      (*gpu_z0, *gpu_count, *gpu_z);
  }

  mandelbrot_count_kernel3<<<dim3(1954U, 1U, 1U), dim3(512U, 1U, 1U)>>>
    (*gpu_count);
  hipMemcpy(&count[0], gpu_count, 8000000ULL, hipMemcpyDeviceToHost);
  hipFree(*gpu_yGrid);
  hipFree(*gpu_xGrid);
  hipFree(*gpu_z);
  hipFree(*gpu_count);
  hipFree(*gpu_z0);
}

void mandelbrot_count_api(const mxArray * const prhs[3], int32_T, const mxArray *
  plhs[1])
{
  real_T (*count)[1000000];
  real_T maxIterations;
  real_T (*xGrid)[1000000];
  real_T (*yGrid)[1000000];
  count = (real_T (*)[1000000])mxMalloc(sizeof(real_T [1000000]));

  // Marshall function inputs
  maxIterations = emlrt_marshallIn(emlrtAliasP(prhs[0]), "maxIterations");
  xGrid = c_emlrt_marshallIn(emlrtAlias(prhs[1]), "xGrid");
  yGrid = c_emlrt_marshallIn(emlrtAlias(prhs[2]), "yGrid");

  // Invoke the target function
  mandelbrot_count(maxIterations, *xGrid, *yGrid, *count);

  // Marshall function outputs
  plhs[0] = emlrt_marshallOut(*count);
}

void mandelbrot_count_atexit()
{
  mexFunctionCreateRootTLS();
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
  emlrtExitTimeCleanup(&emlrtContextGlobal);
}

void mandelbrot_count_initialize()
{
  mex_InitInfAndNan();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, 0);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2012b(emlrtRootTLSGlobal, "Distrib_Computing_Toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
  hipGetLastError();
}

void mandelbrot_count_terminate()
{
  hipError_t errCode;
  errCode = hipGetLastError();
  if (errCode != hipSuccess) {
    emlrtThinCUDAError(false, emlrtRootTLSGlobal);
  }

  emlrtLeaveRtStackR2012b(emlrtRootTLSGlobal);
  emlrtDestroyRootTLS(&emlrtRootTLSGlobal);
}

// End of code generation (mandelbrot_count.cu)
